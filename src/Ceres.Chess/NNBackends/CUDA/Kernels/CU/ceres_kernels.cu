#include "hip/hip_runtime.h"
#region License notice

/*
  This file is part of the Ceres project at https://github.com/dje-dev/ceres.
  Copyright (C) 2020- by David Elliott and the Ceres Authors.

  Ceres is free software under the terms of the GNU General Public License v3.0.
  You should have received a copy of the GNU General Public License
  along with Ceres. If not, see <http://www.gnu.org/licenses/>.
*/

#endregion

#include "cuda_common.h"
namespace ceres {


__global__ void copyMaskedMovesKernel(half* inPolicies, short* inMovesIndices, float* outPoliciesMasked, int size) 
{
  int itemIndex = threadIdx.x + blockDim.x * blockIdx.x;

  int offsetPolicies = 1858 * itemIndex;
  int offsetMoves = 96 * itemIndex;
  int offsetPoliciesOut = 96 * itemIndex;

  if (itemIndex < size)
  {
    for (int i=0;i<96;i++)
    {
      outPoliciesMasked[offsetPoliciesOut + i] = inPolicies[offsetPolicies + inMovesIndices[offsetMoves + i]];
//      outPoliciesMasked[offsetPoliciesOut + i] = exp((float)inPolicies[offsetPolicies + inMovesIndices[offsetMoves + i]]);
    }
  }

  }


__global__ void shiftConvertKernel(half* outV, char* inData, float minVal, float maxVal, int size) 
{
  int itemIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if (itemIndex < size)
  {
    char b0 = inData[itemIndex * 2];
    char b1 = inData[itemIndex * 2 + 1];
    float v1 = 256 * b1 + b0;

    outV[itemIndex] = (half)(minVal + v1 * (maxVal - minVal) / 65535.0f);
  }
}

}